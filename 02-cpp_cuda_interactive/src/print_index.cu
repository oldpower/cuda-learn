#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "utils.hpp"

__global__ void print_idx_kernal()
{
    printf("block idx: (%3d,%3d,%3d), thread inx: (%3d,%3d,%3d) \n",
            blockIdx.z,blockIdx.y,blockIdx.x,
            threadIdx.z,threadIdx.y,threadIdx.x);
}

__global__ void print_dim_kernal()
{
    printf("grid dim: (%3d,%3d,%3d), block dim(%3d,%3d,%3d)\n",
            gridDim.z,gridDim.y,gridDim.x,
            blockDim.z,blockDim.y,blockDim.x);
}
__global__ void print_thread_idx_per_block_kernal()
{
    int index = threadIdx.z + blockDim.x * blockDim.y +
                threadIdx.y + blockDim.x +
                threadIdx.y;
    printf("block idx:(%3d,%3d,%3d),thread idx:%3d\n",
            blockIdx.z,blockIdx.y,blockIdx.x,
            index);
}

__global__ void print_thread_idx_per_grid_kernal()
{
    int bSize = blockDim.z * blockDim.y * blockDim.x;

    int bIndex = blockIdx.z * gridDim.x * gridDim.y +
                blockIdx.y * gridDim.x +
                blockIdx.x;
    int tIndex = threadIdx.z * blockDim.x * blockDim.y +
                threadIdx.y * blockDim.x + 
                threadIdx.x;

    int index = bIndex * bSize + tIndex;

    printf("block idx: %3d, thread idx in block: %3d, thread idx: %3d\n",
            bIndex,tIndex,index);
}

void print_idx_device(dim3 grid,dim3 block)
{
    print_idx_kernal<<<grid,block>>>();
    CUDA_CHECK(hipDeviceSynchronize());
}

void print_dim_device(dim3 gird,dim3 block)
{
    print_dim_kernal<<<gird,block>>>();
    CUDA_CHECK(hipDeviceSynchronize());
}
void print_thread_idx_per_block_device(dim3 grid,dim3 block)
{

    print_thread_idx_per_block_kernal<<<grid,block>>>();
    CUDA_CHECK(hipDeviceSynchronize());
}
void print_thread_idx_device(dim3 grid,dim3 block)
{
    print_thread_idx_per_grid_kernal<<<grid,block>>>();
    CUDA_CHECK(hipDeviceSynchronize());
} 


