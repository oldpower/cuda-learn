#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#include "utils.hpp"


__global__ void MatmulSharedStaticKernel (float *M_device, float *N_device, float *P_device, int width){
    __shared__ float M_deviceShared[BLOCKSIZE][BLOCKSIZE];
    __shared__ float N_deviceShared[BLOCKSIZE][BLOCKSIZE];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float P_element = 0.0;

    int ty = threadIdx.y;
    int tx = threadIdx.x;

    for(int m = 0 ; m < width/BLOCKSIZE ; m++){
    	M_deviceShared[ty][tx] = M_device[y * width + (m * BLOCKSIZE + tx)];
	N_deviceShared[ty][tx] = N_device[(m * BLOCKSIZE + ty) * width + x];

	__syncthreads();

	for(int k = 0 ; k < BLOCKSIZE; k++){
	    P_element += M_deviceShared[ty][k] * N_deviceShared[k][tx];
	}

	__syncthreads();
    }

    P_device[y * width + x] = P_element;

}

__global__ void MatmulSharedDynamicKernel (float *M_device, float *N_device, float *P_device, int width, int blockSize){
    //动态共享内存是一维的，使用时需要extern声明
    extern __shared__ float deviceShared[];

    int stride = blockSize * blockSize;

    int x = blockIdx.x * blockSize + threadIdx.x;
    int y = blockIdx.y * blockSize + threadIdx.y;

    float P_element = 0.0;

    int ty = threadIdx.y;
    int tx = threadIdx.x;

    for(int m = 0; m < width / blockSize; m++){
    	deviceShared[ty * blockSize + tx] = M_device[y * width + (m * blockSize + tx)];
	deviceShared[stride + (ty * blockSize + tx)] = N_device[(m * blockSize + ty) * width + x];

	__syncthreads();

	for(int k = 0; k < blockSize; k++){
	    P_element += deviceShared[ty * blockSize + k] * deviceShared[stride + (k * blockSize + tx)];
	}

	__syncthreads();
    }

    if (y < width && x < width){
    	P_device[y * width + x] = P_element;
    }
}


void MatmulSharedOnDevice(float *M_host,float *N_host, float* P_host, int width,int blockSize, bool staticMem)
{
    //设置矩阵大小
    int size = width * width * sizeof(float);
    //设置共享内存块大小
    long int sMemSize = blockSize * blockSize * sizeof(float) * 2;

    
    //分配GPU空间
    float *M_device;
    float *N_device;
    CUDA_CHECK(hipMalloc(&M_device,size));
    CUDA_CHECK(hipMalloc(&N_device,size));
    float *P_device;
    CUDA_CHECK(hipMalloc(&P_device,size));

    //将host数据copy至GPU
    CUDA_CHECK(hipMemcpy(M_device,M_host,size,hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(N_device,N_host,size,hipMemcpyHostToDevice));

    //核函数matmul
    dim3 dimBlock(blockSize,blockSize);
    dim3 dimGrid(width/blockSize,width/blockSize);

    if (staticMem){
    	MatmulSharedStaticKernel <<<dimGrid,dimBlock>>>(M_device,N_device,P_device,width);
    }else{
	// nullptr 是默认 stream
	MatmulSharedDynamicKernel <<<dimGrid, dimBlock, sMemSize, nullptr>>>(M_device, N_device, P_device, width, blockSize);
    }

    // 将结果copy到host
    CUDA_CHECK(hipMemcpy(P_host,P_device,size,hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());

    // 排查kernal错误
    LAST_KERNEL_CHECK();

    //Free
    hipFree(P_device);
    hipFree(N_device);
    hipFree(M_device);
}
