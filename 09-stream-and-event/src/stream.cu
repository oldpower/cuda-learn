#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include "utils.hpp"

#define MAX_ITER 1000   // memcpy == kernel / 1   (开始能够看出来kernel的overlapping)
// #define MAX_ITER 10000   // memcpy == kernel / 10   (开始能够看出来kernel的overlapping)
// #define MAX_ITER 100000   // memcpy == kernel / 100   (开始能够看出来kernel的overlapping)
#define SIZE 32

// 为了能够体现延迟，这里特意使用clock64()来进行模拟sleep
// 否则如果kernel计算太快，而无法观测到kernel在multi stream中的并发
__global__ void SleepKernel(
    int64_t num_cycles)
{
    int64_t cycles = 0;
    int64_t start = clock64();
    while(cycles < num_cycles){
	cycles = clock64() - start;
    }
}	


/* 1 stream，处理一次memcpy，以及n个kernel */
void SleepSingleStream(
    float* src_host, float* tar_host,
    int width, int blockSize,
    int count)
{
    int size = width * width * sizeof(float);

    float *src_device;
    float *tar_device;

    CUDA_CHECK(hipMalloc((void**)&src_device,size));
    CUDA_CHECK(hipMalloc((void**)&tar_device,size));

    for(int i = 0; i < count; i++){
    	for(int j = 0; j < 1; j++)
	    CUDA_CHECK(hipMemcpy(src_device, src_host, size, hipMemcpyHostToDevice));

	dim3 dimBlock(blockSize, blockSize);
	dim3 dimGrid(width / blockSize, width / blockSize);

	SleepKernel <<<dimGrid, dimBlock>>> (MAX_ITER);

	CUDA_CHECK(hipMemcpy(src_host, src_device, size, hipMemcpyDeviceToHost));
    }

    CUDA_CHECK(hipDeviceSynchronize());
    
    hipFree(tar_device);
    hipFree(src_device);
}

/* n stream，处理一次memcpy，以及n个kernel */
void SleepMultiStream(
    float* src_host, float* tar_host, 
    int width, int blockSize,
    int count)
{
    int size = width * width * sizeof(float);

    float *src_device;
    float *tar_device;
    
    CUDA_CHECK(hipMalloc((void**)&src_device, size));
    CUDA_CHECK(hipMalloc((void**)&tar_device, size));

    /* 先把所需的stream创建出来 */
    hipStream_t stream[count];
    for(int i = 0; i < count ; i++){
        CUDA_CHECK(hipStreamCreate(&stream[i]));
    } 

    for(int i = 0; i < count; i++){
    	for(int j = 0; j < 1; j++)
	    CUDA_CHECK(hipMemcpyAsync(src_device, src_host, size, hipMemcpyHostToDevice, stream[i]));
	
	dim3 dimBlock(blockSize, blockSize);
	dim3 dimGrid(width / blockSize, width / blockSize);

	/* 这里面我们把参数写全了 <<<dimGrid, dimBlock, shareMemSize, stream>>> */
	SleepKernel <<<dimGrid, dimBlock, 0, stream[i]>>> (MAX_ITER);
	CUDA_CHECK(hipMemcpyAsync(src_host, src_device, size, hipMemcpyDeviceToHost, stream[i]));
    }

    CUDA_CHECK(hipDeviceSynchronize());

    hipFree(tar_device);
    hipFree(src_device);

    /* 记得释放steam , 使用 hipStreamDestroy*/
    for(int i = 0; i < count; i++){
    	hipStreamDestroy(stream[i]);
    }

}
