#include "hip/hip_runtime.h"

__global__ void MatmulKernel(float *M_device,float *N_device,float *P_device,int width)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int P_element = 0;

    for (int k=0;k<width;k++)
    {
        float M_element = M_device[width * y + k];
        float N_element = N_device[width * k + x];
        P_element += M_element*N_element;
    }
    P_device[width * y + x] = P_element;
}

void MatmulOnDevice(float *M_host,float *N_host, float* P_host, int width,int blockSize)
{
    //设置矩阵大小
    int size = width * width * sizeof(float);
    
    //分配GPU空间
    float *M_device;
    float *N_device;
    hipMalloc(&M_device,size);
    hipMalloc(&N_device,size);
    float *P_device;
    hipMalloc(&P_device,size);

    //将host数据copy至GPU
    hipMemcpy(M_device,M_host,size,hipMemcpyHostToDevice);
    hipMemcpy(N_device,N_host,size,hipMemcpyHostToDevice);

    //核函数matmul
    dim3 dimBlock(blockSize,blockSize);
    dim3 dimGrid(width/blockSize,width/blockSize);
    MatmulKernel<<<dimGrid,dimBlock>>>(M_device,N_device,P_device,width);

    // 将结果copy到host
    hipMemcpy(P_host,P_device,size,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    //Free
    hipFree(P_device);
    hipFree(N_device);
    hipFree(M_device);
}
