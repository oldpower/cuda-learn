#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#include "utils.hpp"

__global__ void MatmulKernel(float *M_device,float *N_device,float *P_device,int width)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float P_element = 0.0;

    for (int k=0;k<width;k++)
    {
        float M_element = M_device[width * y + k];
        float N_element = N_device[width * k + x];
        P_element += M_element*N_element;
    }
    P_device[width * y + x] = P_element;
}

void MatmulOnDevice(float *M_host,float *N_host, float* P_host, int width,int blockSize)
{
    //设置矩阵大小
    int size = width * width * sizeof(float);
    
    //分配GPU空间
    float *M_device;
    float *N_device;
    CUDA_CHECK(hipMalloc(&M_device,size));
    CUDA_CHECK(hipMalloc(&N_device,size));
    float *P_device;
    CUDA_CHECK(hipMalloc(&P_device,size));

    //将host数据copy至GPU
    CUDA_CHECK(hipMemcpy(M_device,M_host,size,hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(N_device,N_host,size,hipMemcpyHostToDevice));

    //核函数matmul
    dim3 dimBlock(blockSize,blockSize);
    dim3 dimGrid(width/blockSize,width/blockSize);
    MatmulKernel<<<dimGrid,dimBlock>>>(M_device,N_device,P_device,width);

    // 将结果copy到host
    CUDA_CHECK(hipMemcpy(P_host,P_device,size,hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());

    // 排查kernal错误
    LAST_KERNEL_CHECK();

    //Free
    hipFree(P_device);
    hipFree(N_device);
    hipFree(M_device);
}
