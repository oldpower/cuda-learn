#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#include "utils.hpp"


__global__ void MatmulSharedStaticConflictKernel (float *M_device, float *N_device, float *P_device, int width){
    __shared__ float M_deviceShared[BLOCKSIZE][BLOCKSIZE];
    __shared__ float N_deviceShared[BLOCKSIZE][BLOCKSIZE];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float P_element = 0.0;

    int ty = threadIdx.y;
    int tx = threadIdx.x;

    for(int m = 0 ; m < width/BLOCKSIZE ; m++){
    	M_deviceShared[tx][ty] = M_device[x * width + (m * BLOCKSIZE + ty)];
	N_deviceShared[tx][ty] = N_device[(m * BLOCKSIZE + tx) * width + y];

	__syncthreads();

	for(int k = 0 ; k < BLOCKSIZE; k++){
	    P_element += M_deviceShared[tx][k] * N_deviceShared[k][ty];
	}

	__syncthreads();
    }

    P_device[x * width + y] = P_element;

}

__global__ void MatmulSharedDynamicConflictKernel (float *M_device, float *N_device, float *P_device, int width, int blockSize){
    //动态共享内存是一维的，使用时需要extern声明
    extern __shared__ float deviceShared[];

    int stride = blockSize * blockSize;

    int x = blockIdx.x * blockSize + threadIdx.x;
    int y = blockIdx.y * blockSize + threadIdx.y;

    float P_element = 0.0;

    int ty = threadIdx.y;
    int tx = threadIdx.x;

    for(int m = 0; m < width / blockSize; m++){
    	deviceShared[tx * blockSize + ty] = M_device[x * width + (m * blockSize + ty)];
	deviceShared[stride + (tx * blockSize + ty)] = N_device[(m * blockSize + tx) * width + y];

	__syncthreads();

	for(int k = 0; k < blockSize; k++){
	    P_element += deviceShared[tx * blockSize + k] * deviceShared[stride + (k * blockSize + ty)];
	}

	__syncthreads();
    }

    if (y < width && x < width){
    	P_device[x * width + y] = P_element;
    }
}


void MatmulSharedConflictOnDevice(float *M_host,float *N_host, float* P_host, int width,int blockSize, bool staticMem)
{
    //设置矩阵大小
    int size = width * width * sizeof(float);
    //设置共享内存块大小
    long int sMemSize = blockSize * blockSize * sizeof(float) * 2;

    
    //分配GPU空间
    float *M_device;
    float *N_device;
    CUDA_CHECK(hipMalloc(&M_device,size));
    CUDA_CHECK(hipMalloc(&N_device,size));
    float *P_device;
    CUDA_CHECK(hipMalloc(&P_device,size));

    //将host数据copy至GPU
    CUDA_CHECK(hipMemcpy(M_device,M_host,size,hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(N_device,N_host,size,hipMemcpyHostToDevice));

    //核函数matmul
    dim3 dimBlock(blockSize,blockSize);
    dim3 dimGrid(width/blockSize,width/blockSize);

    if (staticMem){
    	MatmulSharedStaticConflictKernel <<<dimGrid,dimBlock>>>(M_device,N_device,P_device,width);
    }else{
	// nullptr 是默认 stream
	MatmulSharedDynamicConflictKernel <<<dimGrid, dimBlock, sMemSize, nullptr>>>(M_device, N_device, P_device, width, blockSize);
    }

    // 将结果copy到host
    CUDA_CHECK(hipMemcpy(P_host,P_device,size,hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());

    // 排查kernal错误
    LAST_KERNEL_CHECK();

    //Free
    hipFree(P_device);
    hipFree(N_device);
    hipFree(M_device);
}
