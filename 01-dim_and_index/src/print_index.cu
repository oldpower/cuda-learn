#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void print_idx_kernal()
{
    printf("block idx: (%3d,%3d,%3d), thread inx: (%3d,%3d,%3d) \n",
            blockIdx.z,blockIdx.y,blockIdx.x,
            threadIdx.z,threadIdx.y,threadIdx.x);
}

__global__ void print_dim_kernal()
{
    printf("grid dim: (%3d,%3d,%3d), block dim(%3d,%3d,%3d)\n",
            gridDim.z,gridDim.y,gridDim.x,
            blockDim.z,blockDim.y,blockDim.x);
}
__global__ void print_thread_idx_per_block_kernal()
{
    int index = threadIdx.z + blockDim.x * blockDim.y +
                threadIdx.y + blockDim.x +
                threadIdx.y;
    printf("block idx:(%3d,%3d,%3d),thread idx:%3d\n",
            blockIdx.z,blockIdx.y,blockIdx.x,
            index);
}

__global__ void print_thread_idx_per_grid_kernal()
{
    int bSize = blockDim.z * blockDim.y * blockDim.x;

    int bIndex = blockIdx.z * gridDim.x * gridDim.y +
                blockIdx.y * gridDim.x +
                blockIdx.x;
    int tIndex = threadIdx.z * blockDim.x * blockDim.y +
                threadIdx.y * blockDim.x + 
                threadIdx.x;

    int index = bIndex * bSize + tIndex;

    printf("block idx: %3d, thread idx in block: %3d, thread idx: %3d\n",
            bIndex,tIndex,index);
}

void print_one_dim()
{
    int inputSize = 8;
    int blockDim = 4;
    int gridDim = inputSize/blockDim;

    dim3 block(blockDim);
    dim3 grid(gridDim);

    // 打印核的索引
    print_idx_kernal<<<grid,block>>>();
    // 打印核的维度
    print_dim_kernal<<<grid,block>>>();
    // 打印核的每个block的thread索引
    print_thread_idx_per_block_kernal<<<grid,block>>>();
    // 打印核的每个grid的thread索引
    print_thread_idx_per_grid_kernal<<<grid,block>>>();

    hipDeviceSynchronize();

}
int main()
{
    print_one_dim(); 

    return 0;
}
